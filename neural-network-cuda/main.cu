#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <cassert>

#include "layers/linear_layer.h"
#include "layers/relu_layer.h"
#include "layers/sigmoid_layer.h"

void print_matrix(const float* data, int rows, int cols) {
    for (int i = 0; i < rows; ++i) {
        for (int j = 0; j < cols; ++j) {
            std::cout << data[i * cols + j] << " ";
        }
        std::cout << "\n";
    }
}

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA Error: " << hipGetErrorString(err) << " at line " << __LINE__ << std::endl; \
        exit(1); \
    } \
} while(0)

void test_linear_backward() {
    const int batch_size = 3;
    const int input_dim = 3;
    const int output_dim = 3;
    const float lr = 0.1f;

    // Create layer
    Linear layer(input_dim, output_dim);

    // Allocate and fill dummy input
    float h_input[batch_size * input_dim] = {
        1.0f, 2.0f, 3.0f,
        4.0f, 5.0f, 6.0f,
        7.0f, 8.0f, 9.0f
    };
    float* d_input;
    CHECK_CUDA(hipMalloc(&d_input, sizeof(float) * batch_size * input_dim));
    CHECK_CUDA(hipMemcpy(d_input, h_input, sizeof(float) * batch_size * input_dim, hipMemcpyHostToDevice));

    // Forward pass
    float* d_output;
    CHECK_CUDA(hipMalloc(&d_output, sizeof(float) * batch_size * output_dim));
    layer.forward(d_input, batch_size);

    // Allocate dummy output gradient (e.g., from MSE loss w.r.t. output)
    float h_dY[batch_size * output_dim] = {
        0.1f, -0.2f,
        0.05f, 0.3f
    };
    float* d_dY;
    CHECK_CUDA(hipMalloc(&d_dY, sizeof(float) * batch_size * output_dim));
    CHECK_CUDA(hipMemcpy(d_dY, h_dY, sizeof(float) * batch_size * output_dim, hipMemcpyHostToDevice));

    // Input gradient output
    float* d_dX;
    CHECK_CUDA(hipMalloc(&d_dX, sizeof(float) * batch_size * input_dim));

    // Store original weights and biases
    float h_W_before[input_dim * output_dim], h_b_before[output_dim];
    CHECK_CUDA(hipMemcpy(h_W_before, layer.d_weights, sizeof(float) * input_dim * output_dim, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_b_before, layer.d_bias, sizeof(float) * output_dim, hipMemcpyDeviceToHost));

    // Backward pass
    layer.backward(d_dY, d_dX, lr, batch_size);

    // Get updated weights and biases
    float h_W_after[input_dim * output_dim], h_b_after[output_dim];
    CHECK_CUDA(hipMemcpy(h_W_after, layer.d_weights, sizeof(float) * input_dim * output_dim, hipMemcpyDeviceToHost));
    CHECK_CUDA(hipMemcpy(h_b_after, layer.d_bias, sizeof(float) * output_dim, hipMemcpyDeviceToHost));

    // Print diffs
    std::cout << "Weight updates:\n";
    for (int i = 0; i < input_dim * output_dim; ++i)
        std::cout << "W[" << i << "]: " << h_W_before[i] << " -> " << h_W_after[i] << "\n";

    std::cout << "Bias updates:\n";
    for (int i = 0; i < output_dim; ++i)
        std::cout << "b[" << i << "]: " << h_b_before[i] << " -> " << h_b_after[i] << "\n";

    // Check input gradient is not all zero
    float h_dX[batch_size * input_dim];
    CHECK_CUDA(hipMemcpy(h_dX, d_dX, sizeof(h_dX), hipMemcpyDeviceToHost));

    bool non_zero = false;
    for (int i = 0; i < batch_size * input_dim; ++i) {
        if (std::abs(h_dX[i]) > 1e-6) {
            non_zero = true;
            break;
        }
    }

    assert(non_zero && "Input gradient is all zeros — backward may have failed!");

    std::cout << "Backward pass test passed.\n";

    // Cleanup
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_dY);
    hipFree(d_dX);
}

void test_relu_backward() 
{
    const int batch_size = 2;
    const int dim = 4;
    const int size = batch_size * dim;

    // Host input
    float h_input[size] = { -1.0f, 2.0f, -3.0f, 4.0f,
                             5.0f, -6.0f, 7.0f, -8.0f };

    float h_grad_out[size] = { 1.0f, 1.0f, 1.0f, 1.0f,
                               1.0f, 1.0f, 1.0f, 1.0f };

    float h_output[size], h_grad_input[size];

    // Device memory
    float* d_input, * d_grad_out, * d_grad_input;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_grad_out, size * sizeof(float));
    hipMalloc(&d_grad_input, size * sizeof(float));

    // Copy input to device
    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_grad_out, h_grad_out, size * sizeof(float), hipMemcpyHostToDevice);

    // Create and run ReLU
    ReLU relu(dim);
    relu.forward(d_input, batch_size);
    float* d_output = relu.get_output();

    // Copy result back
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "ReLU forward output:\n";
    print_matrix(h_output, 1, size);

    // Run backward
    relu.backward(d_grad_out, d_grad_input, 0.0f, batch_size);
    hipMemcpy(h_grad_input, d_grad_input, size * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "ReLU backward grad_input:\n";
    print_matrix(h_grad_input, 1, size);

    // Free
    hipFree(d_input);
    hipFree(d_grad_out);
    hipFree(d_grad_input);
}

void test_sigmoid_backward()
{
    const int batch_size = 2;
    const int dim = 4;
    const int size = batch_size * dim;

    float h_input[size] = {
        -2.0f, 0.0f, 1.0f, 2.0f,
         3.0f, -1.0f, -4.0f, 0.5f
    };

    float h_output_grad[size] = {
        1.0f, 1.0f, 1.0f, 1.0f,
        1.0f, 1.0f, 1.0f, 1.0f
    };

    // Allocate device memory
    float* d_input, * d_output_grad, * d_input_grad;
    hipMalloc(&d_input, size * sizeof(float));
    hipMalloc(&d_output_grad, size * sizeof(float));
    hipMalloc(&d_input_grad, size * sizeof(float));

    hipMemcpy(d_input, h_input, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_output_grad, h_output_grad, size * sizeof(float), hipMemcpyHostToDevice);

    // Initialize sigmoid layer
    Sigmoid sigmoid(dim);

    // Forward
    sigmoid.forward(d_input, batch_size);
    float* d_output = sigmoid.get_output();
    float h_output[size];
    hipMemcpy(h_output, d_output, size * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Sigmoid Forward Output: \n";
    print_matrix(h_output, batch_size, dim);

    // Backward
    sigmoid.backward(d_output_grad, d_input_grad, 0.0f, batch_size);
    float h_input_grad[size];
    hipMemcpy(h_input_grad, d_input_grad, size * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << "Sigmoid Backward d_input_grad: \n";
    print_matrix(h_input_grad, batch_size, dim);

    // Cleanup
    hipFree(d_input);
    hipFree(d_output_grad);
    hipFree(d_input_grad);
}

int main()
{
    test_linear_backward();
    test_relu_backward();
    test_sigmoid_backward();

    //TODO: Implement Binary Cross Entropy Loss

    return 0;
}