﻿#include <iostream>

#include "layers/linear_layer.h"
#include "layers/relu_layer.h"
#include "layers/sigmoid_layer.h"
#include "layers/sequential.h"
#include "loss/bce_loss.h"
#include "accuracy/bin_acc.h"
#include "utils/utils.h"
#include "train/train.h"
#include "data/data_load.h"

int main()
{
    int input_dim = 64 * 64;
    int hidden_dim = 1024;
    int output_dim = 1;

    std::vector<float> vec_train_inputs;
    std::vector<float> vec_train_labels;

    if (!load_csv_data("data/train_data.csv", vec_train_inputs, vec_train_labels, input_dim))
    {
        std::cerr << "Failed to load train data!" << std::endl;
        return -1;
    }

    normalize_data(vec_train_inputs);

    unsigned int num_samples_train = static_cast<int>(vec_train_labels.size());
    unsigned int batch_size = 10;
    const float learning_rate = 0.01f;
    const int epoches = 150;

    float* h_train_inputs = vec_train_inputs.data();
    float* h_train_labels = vec_train_labels.data();

    Sequential model;
    model.add_layer(new Linear(input_dim, hidden_dim));
    model.add_layer(new ReLU(hidden_dim));
    model.add_layer(new Linear(hidden_dim, output_dim));
    model.add_layer(new Sigmoid(output_dim));

    BCELoss loss_fn;
    BinaryAccuracy acc_fn(0.5);

    train_model(model, loss_fn, acc_fn, h_train_inputs, h_train_labels, num_samples_train, input_dim, batch_size, epoches, learning_rate, true);

    std::vector<float> vec_val_inputs;
    std::vector<float> vec_val_labels;

    if (!load_csv_data("data/val_data.csv", vec_val_inputs, vec_val_labels, input_dim))
    {
        std::cerr << "Failed to load validate data!" << std::endl;
        return -1;
    }

    int num_samples_val = static_cast<int>(vec_val_labels.size());

    return 0;

    //TODO: Add test data routine
}