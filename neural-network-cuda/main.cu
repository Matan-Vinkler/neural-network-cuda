﻿#include <hip/hip_runtime.h>
#include <>

#include <iostream>
#include <cassert>

#include "layers/linear_layer.h"
#include "layers/relu_layer.h"
#include "layers/sigmoid_layer.h"
#include "layers/sequential.h"
#include "loss/bce_loss.h"

#include "utils/utils.h"

#include "train/train.h"

void test_train_model()
{
    const int input_dim = 2;
    const int hidden_dim = 3;
    const int output_dim = 1;
    const int batch_size = 4;
    const int num_samples = 4;
    const float learning_rate = 0.1f;
    const int epoches = 1000;

    float h_input[batch_size * input_dim] = {
        1.0f, 2.0f,
        2.0f, 1.0f,
        0.0f, 1.0f,
        1.0f, 1.0f
    };

    float h_labels[batch_size] = { 1, 0, 0, 1 };

    Sequential model;
    model.add_layer(new Linear(input_dim, hidden_dim, true));
    model.add_layer(new ReLU(hidden_dim));
    model.add_layer(new Linear(hidden_dim, output_dim));
    model.add_layer(new Sigmoid(output_dim));

    BCELoss loss;

    train_model(model, loss, h_input, h_labels, num_samples, input_dim, batch_size, epoches, learning_rate, true);
}

int main()
{
    for (int i = 0; i < 1; i++)
    {
        std::cout << "[---------------------- Test " << i + 1 << " ----------------------]\n";
        test_train_model();
        std::cout << "[----------------------------------------------------]\n\n";
    }

    std::cout << std::endl;

    //TODO: Fix loss stuck value (maybe gradient vanishing)
    //TODO: Implement data load

    return 0;
}